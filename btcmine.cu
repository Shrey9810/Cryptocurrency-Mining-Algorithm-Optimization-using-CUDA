// Compile with: nvcc btcmine.cu -lnvml

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <time.h>
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>

// ---------------------------------------------------------------------------
// Minimal SHA-256 Implementation for Host (used for the merkle root)
// ---------------------------------------------------------------------------
#define SHA256_BLOCK_SIZE 32  // SHA256 outputs 32 bytes

typedef struct {
    uint8_t data[64];
    uint32_t datalen;
    unsigned long long bitlen;
    uint32_t state[8];
} SHA256_CTX;

static const uint32_t k_host[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

void sha256_transform(SHA256_CTX *ctx, const uint8_t data[])
{
    uint32_t a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) |
               (data[j + 2] << 8) | (data[j + 3]);
    for ( ; i < 64; ++i) {
        uint32_t s0 = ((m[i - 15] >> 7) | (m[i - 15] << (32 - 7))) ^
                      ((m[i - 15] >> 18) | (m[i - 15] << (32 - 18))) ^
                      (m[i - 15] >> 3);
        uint32_t s1 = ((m[i - 2] >> 17) | (m[i - 2] << (32 - 17))) ^
                      ((m[i - 2] >> 19) | (m[i - 2] << (32 - 19))) ^
                      (m[i - 2] >> 10);
        m[i] = m[i - 16] + s0 + m[i - 7] + s1;
    }

    a = ctx->state[0];
    b = ctx->state[1];
    c = ctx->state[2];
    d = ctx->state[3];
    e = ctx->state[4];
    f = ctx->state[5];
    g = ctx->state[6];
    h = ctx->state[7];

    for (i = 0; i < 64; ++i) {
        uint32_t S1 = ((e >> 6) | (e << (32 - 6))) ^
                      ((e >> 11) | (e << (32 - 11))) ^
                      ((e >> 25) | (e << (32 - 25)));
        uint32_t ch = (e & f) ^ ((~e) & g);
        t1 = h + S1 + ch + k_host[i] + m[i];
        uint32_t S0 = ((a >> 2) | (a << (32 - 2))) ^
                      ((a >> 13) | (a << (32 - 13))) ^
                      ((a >> 22) | (a << (32 - 22)));
        uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
        t2 = S0 + maj;

        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    ctx->state[0] += a;
    ctx->state[1] += b;
    ctx->state[2] += c;
    ctx->state[3] += d;
    ctx->state[4] += e;
    ctx->state[5] += f;
    ctx->state[6] += g;
    ctx->state[7] += h;
}

void sha256_init(SHA256_CTX *ctx)
{
    ctx->datalen = 0;
    ctx->bitlen = 0;
    ctx->state[0] = 0x6a09e667;
    ctx->state[1] = 0xbb67ae85;
    ctx->state[2] = 0x3c6ef372;
    ctx->state[3] = 0xa54ff53a;
    ctx->state[4] = 0x510e527f;
    ctx->state[5] = 0x9b05688c;
    ctx->state[6] = 0x1f83d9ab;
    ctx->state[7] = 0x5be0cd19;
}

void sha256_update(SHA256_CTX *ctx, const uint8_t data[], size_t len)
{
    for (size_t i = 0; i < len; i++) {
        ctx->data[ctx->datalen] = data[i];
        ctx->datalen++;
        if (ctx->datalen == 64) {
            sha256_transform(ctx, ctx->data);
            ctx->bitlen += 512;
            ctx->datalen = 0;
        }
    }
}

void sha256_final(SHA256_CTX *ctx, uint8_t hash[])
{
    uint32_t i = ctx->datalen;

    if (ctx->datalen < 56) {
        ctx->data[i++] = 0x80;
        while (i < 56)
            ctx->data[i++] = 0x00;
    } else {
        ctx->data[i++] = 0x80;
        while (i < 64)
            ctx->data[i++] = 0x00;
        sha256_transform(ctx, ctx->data);
        memset(ctx->data, 0, 56);
    }

    ctx->bitlen += ctx->datalen * 8;
    ctx->data[63] = ctx->bitlen;
    ctx->data[62] = ctx->bitlen >> 8;
    ctx->data[61] = ctx->bitlen >> 16;
    ctx->data[60] = ctx->bitlen >> 24;
    ctx->data[59] = ctx->bitlen >> 32;
    ctx->data[58] = ctx->bitlen >> 40;
    ctx->data[57] = ctx->bitlen >> 48;
    ctx->data[56] = ctx->bitlen >> 56;
    sha256_transform(ctx, ctx->data);

    for (i = 0; i < 4; ++i) {
        hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0xff;
        hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0xff;
        hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0xff;
        hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0xff;
        hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0xff;
        hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0xff;
        hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0xff;
        hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0xff;
    }
}

void sha256_hash(const uint8_t *data, size_t len, uint8_t hash[SHA256_BLOCK_SIZE])
{
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, data, len);
    sha256_final(&ctx, hash);
}

// ---------------------------------------------------------------------------
// Device SHA-256 Implementation (actual work in the kernel)
// ---------------------------------------------------------------------------

__device__ const uint32_t k_device[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

typedef struct {
    uint8_t data[64];
    uint32_t datalen;
    unsigned long long bitlen;
    uint32_t state[8];
} SHA256_CTX_DEV;

__device__ void sha256_init_device(SHA256_CTX_DEV *ctx) {
    ctx->datalen = 0;
    ctx->bitlen = 0;
    ctx->state[0] = 0x6a09e667;
    ctx->state[1] = 0xbb67ae85;
    ctx->state[2] = 0x3c6ef372;
    ctx->state[3] = 0xa54ff53a;
    ctx->state[4] = 0x510e527f;
    ctx->state[5] = 0x9b05688c;
    ctx->state[6] = 0x1f83d9ab;
    ctx->state[7] = 0x5be0cd19;
}

__device__ void sha256_transform_device(SHA256_CTX_DEV *ctx, const uint8_t data[64]) {
    uint32_t a, b, c, d, e, f, g, h;
    uint32_t t1, t2, m[64];
    int i, j;

    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) |
               (data[j + 2] << 8) | (data[j + 3]);
    for ( ; i < 64; ++i) {
        uint32_t s0 = ((m[i - 15] >> 7) | (m[i - 15] << (32 - 7))) ^
                      ((m[i - 15] >> 18) | (m[i - 15] << (32 - 18))) ^
                      (m[i - 15] >> 3);
        uint32_t s1 = ((m[i - 2] >> 17) | (m[i - 2] << (32 - 17))) ^
                      ((m[i - 2] >> 19) | (m[i - 2] << (32 - 19))) ^
                      (m[i - 2] >> 10);
        m[i] = m[i - 16] + s0 + m[i - 7] + s1;
    }

    a = ctx->state[0];
    b = ctx->state[1];
    c = ctx->state[2];
    d = ctx->state[3];
    e = ctx->state[4];
    f = ctx->state[5];
    g = ctx->state[6];
    h = ctx->state[7];

    for (i = 0; i < 64; ++i) {
        uint32_t S1 = ((e >> 6) | (e << (32 - 6))) ^
                      ((e >> 11) | (e << (32 - 11))) ^
                      ((e >> 25) | (e << (32 - 25)));
        uint32_t ch = (e & f) ^ ((~e) & g);
        t1 = h + S1 + ch + k_device[i] + m[i];
        uint32_t S0 = ((a >> 2) | (a << (32 - 2))) ^
                      ((a >> 13) | (a << (32 - 13))) ^
                      ((a >> 22) | (a << (32 - 22)));
        uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
        t2 = S0 + maj;
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    ctx->state[0] += a;
    ctx->state[1] += b;
    ctx->state[2] += c;
    ctx->state[3] += d;
    ctx->state[4] += e;
    ctx->state[5] += f;
    ctx->state[6] += g;
    ctx->state[7] += h;
}

__device__ void sha256_update_device(SHA256_CTX_DEV *ctx, const uint8_t data[], size_t len) {
    for (size_t i = 0; i < len; i++) {
        ctx->data[ctx->datalen] = data[i];
        ctx->datalen++;
        if (ctx->datalen == 64) {
            sha256_transform_device(ctx, ctx->data);
            ctx->bitlen += 512;
            ctx->datalen = 0;
        }
    }
}

__device__ void sha256_final_device(SHA256_CTX_DEV *ctx, uint8_t hash[32]) {
    uint32_t i = ctx->datalen;

    if (ctx->datalen < 56) {
        ctx->data[i++] = 0x80;
        while (i < 56)
            ctx->data[i++] = 0x00;
    } else {
        ctx->data[i++] = 0x80;
        while (i < 64)
            ctx->data[i++] = 0x00;
        sha256_transform_device(ctx, ctx->data);
        for (i = 0; i < 56; i++)
            ctx->data[i] = 0;
    }

    ctx->bitlen += ctx->datalen * 8;
    ctx->data[63] = ctx->bitlen;
    ctx->data[62] = ctx->bitlen >> 8;
    ctx->data[61] = ctx->bitlen >> 16;
    ctx->data[60] = ctx->bitlen >> 24;
    ctx->data[59] = ctx->bitlen >> 32;
    ctx->data[58] = ctx->bitlen >> 40;
    ctx->data[57] = ctx->bitlen >> 48;
    ctx->data[56] = ctx->bitlen >> 56;
    sha256_transform_device(ctx, ctx->data);

    for (i = 0; i < 4; ++i) {
        hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0xff;
        hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0xff;
        hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0xff;
        hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0xff;
        hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0xff;
        hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0xff;
        hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0xff;
        hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0xff;
    }
}

__device__ void sha256_device(const uint8_t *data, size_t len, uint8_t hash[32]) {
    SHA256_CTX_DEV ctx;
    sha256_init_device(&ctx);
    sha256_update_device(&ctx, data, len);
    sha256_final_device(&ctx, hash);
}

// Packs the BlockHeader fields into an 80-byte array (big-endian) and performs double SHA-256.
typedef struct {
    uint32_t version;
    uint8_t prevBlock[32];
    uint8_t merkleRoot[32];
    uint32_t time;
    uint32_t bits;
    uint32_t nonce; // This is varied by each thread.
} BlockHeader;

__device__ void pack_uint32_be(uint32_t val, uint8_t out[4]) {
    out[0] = (uint8_t)((val >> 24) & 0xff);
    out[1] = (uint8_t)((val >> 16) & 0xff);
    out[2] = (uint8_t)((val >> 8) & 0xff);
    out[3] = (uint8_t)(val & 0xff);
}

__device__ void double_sha256_device(const BlockHeader *header, uint8_t hash_out[32]) {
    uint8_t header_bytes[80];
    // Pack version.
    pack_uint32_be(header->version, header_bytes);
    // Copy prevBlock.
    for (int i = 0; i < 32; i++)
        header_bytes[4 + i] = header->prevBlock[i];
    // Copy merkleRoot.
    for (int i = 0; i < 32; i++)
        header_bytes[36 + i] = header->merkleRoot[i];
    // Pack time.
    pack_uint32_be(header->time, header_bytes + 68);
    // Pack bits.
    pack_uint32_be(header->bits, header_bytes + 72);
    // Pack nonce.
    pack_uint32_be(header->nonce, header_bytes + 76);

    uint8_t first_hash[32];
    sha256_device(header_bytes, 80, first_hash);
    sha256_device(first_hash, 32, hash_out);
}

// ---------------------------------------------------------------------------
// Kernel and mining code
// ---------------------------------------------------------------------------

// Adjust these grid parameters for your GPU.
#define NUM_BLOCKS 10240
#define NUM_THREADS 1024
#define TOTAL_THREADS (NUM_BLOCKS * NUM_THREADS)

// Fixed difficulty target: first two bytes must be zero.
__constant__ uint8_t target[32] = {
    0x00, 0x00, 0xFF, 0xFF, 
    0xFF, 0xFF, 0xFF, 0xFF,
    0xFF, 0xFF, 0xFF, 0xFF,
    0xFF, 0xFF, 0xFF, 0xFF,
    0xFF, 0xFF, 0xFF, 0xFF,
    0xFF, 0xFF, 0xFF, 0xFF,
    0xFF, 0xFF, 0xFF, 0xFF,
    0xFF, 0xFF, 0xFF, 0xFF
};

// Device constant memory for the computed merkle root and previous block hash
__constant__ uint8_t d_merkleRoot[32];
__constant__ uint8_t d_prevBlockHash[32];

// Structure to hold each thread's computed nonce and hash.
typedef struct {
    uint32_t nonce;
    uint8_t hash[32];
} Result;

// Compare two 32-byte arrays.
__device__ bool isValidHash(const uint8_t *hash, const uint8_t *target_val) {
    for (int i = 0; i < 32; i++) {
        if (hash[i] < target_val[i])
            return true;
        else if (hash[i] > target_val[i])
            return false;
    }
    return false;
}

// Kernel: performs actual double SHA-256 work and updates a counter of bytes processed.
__global__ void mine_kernel(Result *results, uint32_t *found_nonce, uint8_t *found_hash, uint32_t current_time, unsigned long long *d_total_bytes) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t nonce = idx;

    BlockHeader header;
    header.version = 0x20000000;
    // Copy previous block hash from constant memory
    for (int i = 0; i < 32; i++)
        header.prevBlock[i] = d_prevBlockHash[i];
    // Copy computed merkle root from constant memory.
    for (int i = 0; i < 32; i++)
        header.merkleRoot[i] = d_merkleRoot[i];
    header.time = current_time;
    header.bits = 0x1d00ffff;  // Fixed difficulty
    header.nonce = nonce;

    uint8_t hash[32];
    double_sha256_device(&header, hash);

    results[idx].nonce = nonce;
    for (int i = 0; i < 32; i++)
        results[idx].hash[i] = hash[i];

    // Update counter: each thread processed 192 bytes (80-byte header becomes 128 bytes after padding in first SHA256,
    // and 32-byte hash padded to 64 bytes in the second SHA256).
    atomicAdd(d_total_bytes, (unsigned long long)192);

    if (isValidHash(hash, target)) {
        uint32_t old = atomicMin(found_nonce, nonce);
        if (nonce < old) {
            for (int i = 0; i < 32; i++)
                found_hash[i] = hash[i];
        }
    }
}

// Structure to represent a transaction
typedef struct {
    char sender[256];
    char receiver[256];
    double amount;
} Transaction;

// Structure to represent a mined block
typedef struct {
    int height;
    uint32_t nonce;
    uint8_t hash[32];
    uint8_t prev_hash[32];
    uint8_t merkle_root[32];
    uint8_t target[32];
    uint32_t timestamp;
    size_t tx_count;
} MinedBlock;

// Function to read transactions from a file
std::vector<Transaction> read_transactions_from_file(const char* filename) {
    std::vector<Transaction> transactions;
    std::ifstream file(filename);
    
    if (!file.is_open()) {
        printf("Error opening file: %s\n", filename);
        return transactions;
    }
    
    std::string line;
    while (std::getline(file, line)) {
        // Skip empty lines and comments
        if (line.empty() || line[0] == '#') continue;
        
        std::istringstream iss(line);
        Transaction tx;
        
        if (iss >> tx.sender >> tx.receiver >> tx.amount) {
            transactions.push_back(tx);
        } else {
            printf("Warning: Could not parse line: %s\n", line.c_str());
        }
    }
    
    file.close();
    return transactions;
}

// Function to build Merkle root from multiple transactions
void build_merkle_root(const std::vector<Transaction>& transactions, uint8_t merkle_root[32]) {
    // For simplicity, we'll just concatenate all transaction data and hash it
    // In a real implementation, you would build a proper Merkle tree
    std::string all_transactions;
    for (const auto& tx : transactions) {
        char buffer[1024];
        snprintf(buffer, sizeof(buffer), "%s|%s|%.8f", tx.sender, tx.receiver, tx.amount);
        all_transactions += buffer;
    }
    
    sha256_hash((const uint8_t*)all_transactions.c_str(), all_transactions.size(), merkle_root);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s <transactions_file>\n", argv[0]);
        return 1;
    }
    
    // Initialize NVML for power usage monitoring.
    nvmlInit();
    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);

    // Initialize previous block hash (genesis block)
    uint8_t prev_block_hash[32] = {0};
    
    // Read all transactions from file
    std::vector<Transaction> all_transactions = read_transactions_from_file(argv[1]);
    if (all_transactions.empty()) {
        printf("No transactions found in file or error reading file.\n");
        nvmlShutdown();
        return 1;
    }
    
    printf("\nTotal transactions to be mined: %zu\n", all_transactions.size());
    
    // Vector to store all mined blocks
    std::vector<MinedBlock> blockchain;
    
    // Process transactions in blocks of 100
    size_t total_blocks = (all_transactions.size() + 99) / 100; // Round up
    size_t tx_index = 0;
    int block_height = 0;
    
    while (tx_index < all_transactions.size()) {
        // Get next 100 transactions (or remaining if less than 100)
        size_t block_tx_count = std::min((size_t)100, all_transactions.size() - tx_index);
        std::vector<Transaction> block_transactions(all_transactions.begin() + tx_index, 
                                                   all_transactions.begin() + tx_index + block_tx_count);
        tx_index += block_tx_count;
        
        printf("\nMining Block %d (%zu transactions)\n", block_height, block_tx_count);
        printf("Transactions in this block:\n");
        for (const auto& tx : block_transactions) {
            printf("- %s -> %s (%.8f BTC)\n", tx.sender, tx.receiver, tx.amount);
        }
        
        // Build Merkle root from transactions
        uint8_t h_merkleRoot[32];
        build_merkle_root(block_transactions, h_merkleRoot);
        
        // Copy the computed merkle root and previous block hash to device constant memory.
        hipMemcpyToSymbol(HIP_SYMBOL(d_merkleRoot), h_merkleRoot, 32 * sizeof(uint8_t));
        hipMemcpyToSymbol(HIP_SYMBOL(d_prevBlockHash), prev_block_hash, 32 * sizeof(uint8_t));

        // Allocate memory for found nonce and hash.
        uint32_t *d_found_nonce;
        uint8_t *d_found_hash;
        uint32_t h_found_nonce = 0xFFFFFFFF;
        uint8_t h_found_hash[32] = {0};

        hipMalloc((void **)&d_found_nonce, sizeof(uint32_t));
        hipMalloc((void **)&d_found_hash, 32 * sizeof(uint8_t));
        hipMemcpy(d_found_nonce, &h_found_nonce, sizeof(uint32_t), hipMemcpyHostToDevice);

        // Allocate memory for kernel results.
        Result *d_results;
        hipMalloc((void **)&d_results, TOTAL_THREADS * sizeof(Result));

        // Allocate and initialize counter for total bytes processed.
        unsigned long long *d_total_bytes;
        unsigned long long h_total_bytes = 0;
        hipMalloc((void **)&d_total_bytes, sizeof(unsigned long long));
        hipMemcpy(d_total_bytes, &h_total_bytes, sizeof(unsigned long long), hipMemcpyHostToDevice);

        // Reset values for new mining attempt
        h_found_nonce = 0xFFFFFFFF;
        hipMemcpy(d_found_nonce, &h_found_nonce, sizeof(uint32_t), hipMemcpyHostToDevice);
        h_total_bytes = 0;
        hipMemcpy(d_total_bytes, &h_total_bytes, sizeof(unsigned long long), hipMemcpyHostToDevice);

        // Get power usage before mining
        unsigned int power_before;
        nvmlDeviceGetPowerUsage(device, &power_before);

        // Create and record CUDA events for timing.
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        // Get current real-time timestamp.
        uint32_t current_time = (uint32_t) time(NULL);

        // Launch kernel.
        mine_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_results, d_found_nonce, d_found_hash, current_time, d_total_bytes);

        // Synchronize to ensure proper timing
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // Get power usage after mining
        unsigned int power_after;
        nvmlDeviceGetPowerUsage(device, &power_after);

        // Calculate elapsed time in seconds.
        float elapsed_ms = 0.0f;
        hipEventElapsedTime(&elapsed_ms, start, stop);
        double elapsed_sec = elapsed_ms / 1000.0;

        // Retrieve the total bytes processed.
        hipMemcpy(&h_total_bytes, d_total_bytes, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        double throughput = (double)h_total_bytes / (1024.0 * 1024.0 * 1024.0) / elapsed_sec;

        // Calculate and print total hashes computed and hash rate.
        unsigned long long total_hashes = TOTAL_THREADS;
        double hash_rate = total_hashes / elapsed_sec;
        
        printf("\nMining Performance Metrics for Block %d:\n", block_height);
        printf("------------------------------------\n");
        printf("Kernel execution time: %.2f ms\n", elapsed_ms);
        printf("Throughput: %.2f GB/s\n", throughput);
        printf("Total Hashes Calculated: %llu\n", total_hashes);
        printf("Hash Rate: %.2f hashes/sec\n", hash_rate);
        printf("Average power: %.2f W\n", (float)(power_after + power_before) / 2000.0f);

        // Retrieve kernel results.
        Result *h_results = new Result[TOTAL_THREADS];
        hipMemcpy(h_results, d_results, TOTAL_THREADS * sizeof(Result), hipMemcpyDeviceToHost);
        hipMemcpy(&h_found_nonce, d_found_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
        hipMemcpy(h_found_hash, d_found_hash, 32 * sizeof(uint8_t), hipMemcpyDeviceToHost);

        if (h_found_nonce != 0xFFFFFFFF) {
            printf("\nBlock %d Mined Successfully!\n", block_height);
            printf("---------------------------\n");
            printf("Nonce: %u\n", h_found_nonce);
            printf("Block Hash: ");
            for (int i = 0; i < 32; i++)
                printf("%02x", h_found_hash[i]);
            printf("\n");
            
            // Store the mined block
            MinedBlock block;
            uint8_t h_target[32];
            block.height = block_height;
            block.nonce = h_found_nonce;
            memcpy(block.hash, h_found_hash, 32);
            memcpy(block.prev_hash, prev_block_hash, 32);
            memcpy(block.merkle_root, h_merkleRoot, 32);
            hipMemcpyFromSymbol(h_target, HIP_SYMBOL(target), 32 * sizeof(uint8_t), 0, hipMemcpyDeviceToHost);
            memcpy(block.target, h_target, 32);
            block.timestamp = current_time;
            block.tx_count = block_tx_count;
            blockchain.push_back(block);
            
            // Update previous block hash for next block
            memcpy(prev_block_hash, h_found_hash, 32);
            block_height++;
        } else {
            printf("\nNo valid nonce found for Block %d. Try again with more hashes.\n", block_height);
            // Clean up and exit if we can't mine a block
            delete[] h_results;
            hipFree(d_found_nonce);
            hipFree(d_found_hash);
            hipFree(d_results);
            hipFree(d_total_bytes);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            nvmlShutdown();
            return 1;
        }

        // Clean up.
        delete[] h_results;
        hipFree(d_found_nonce);
        hipFree(d_found_hash);
        hipFree(d_results);
        hipFree(d_total_bytes);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Print the entire blockchain at the end
    printf("\n\nFinal Blockchain:\n");
    printf("=================\n");
    for (const auto& block : blockchain) {
        printf("\nBlock %d:\n", block.height);
        printf("  Hash: ");
        for (int i = 0; i < 32; i++) printf("%02x", block.hash[i]);
        printf("\n");
        printf("  Previous Hash: ");
        for (int i = 0; i < 32; i++) printf("%02x", block.prev_hash[i]);
        printf("\n");
        printf("  Target: ");
        for (int i = 0; i < 32; i++) printf("%02x", block.target[i]);  // <-- Print target
        printf("\n");
        printf("  Merkle Root: ");
        for (int i = 0; i < 32; i++) printf("%02x", block.merkle_root[i]);
        printf("\n");
        printf("  Timestamp: %u\n", block.timestamp);
        printf("  Nonce: %u\n", block.nonce);
    }

    nvmlShutdown();
    return 0;
}